#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Kernels








// Class methods
Simulator::Simulator() {
    numberOfCircles = 0;
    position = NULL;
    velocity = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
}

Simulator::~Simulator() {
    if (position) {
        delete [] position;
        delete [] velocity;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
    }
}

const float*
Simulator::getPosition() {

    printf("Copying position data from device\n");

    // Have host_positions as a class attribute which you memcpy into
    float *host_positions = (float *)malloc(sizeof(float) * 3 * numberOfCircles);

    hipMemcpy(host_positions,
               cudaDevicePosition,
               sizeof(float) * 3 * numberOfCircles,
               hipMemcpyDeviceToHost);

    return host_positions;
}