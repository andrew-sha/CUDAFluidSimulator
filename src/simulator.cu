#include "hip/hip_runtime.h"
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <chrono>
#include <iomanip>
#include <iostream>
#include <stdio.h>

#include "simulator.h"

#define MAX_THREADS_PER_BLOCK (128)
#define PUSH_STRENGTH (5.f)
#define CHUNK_COUNT (1)
#define EPS_F (1e-4f)

extern bool mouseClicked;
extern int2 clickCoords;

__constant__ Settings deviceSettings;

// Return 3D coordinates of neighbor grid cell a particle belongs to
__device__ int3 getGridCell(float3 position) {
    int3 gridCell;
    gridCell.x = (int)(position.x / deviceSettings.h);
    if (gridCell.x < 0 || gridCell.x >= deviceSettings.numCellsPerDim) {
        printf("OOB particle: x = %d\n", gridCell.x);
        printf("(%f, %f, %f)\n", position.x, position.y, position.z);
    }
    gridCell.y = (int)(position.y / deviceSettings.h);
    if (gridCell.y < 0 || gridCell.y >= deviceSettings.numCellsPerDim) {
        printf("OOB particle: y = %d\n", gridCell.y);
        printf("(%f, %f, %f)\n", position.x, position.y, position.z);
    }
    gridCell.z = (int)(position.z / deviceSettings.h);
    if (gridCell.z < 0 || gridCell.z >= deviceSettings.numCellsPerDim) {
        printf("OOB particle: z = %d\n", gridCell.z);
        printf("(%f, %f, %f)\n", position.x, position.y, position.z);
    }

    return gridCell;
}

// Convert 3D coordinates of neighbor grid cell to corresponding array index
__device__ int flattenGridCoord(int3 coord) {
    return coord.x + coord.y * deviceSettings.numCellsPerDim +
           coord.z * deviceSettings.numCellsPerDim *
               deviceSettings.numCellsPerDim;
}

// Smoothing kernel for density updates
__device__ float densityKernel(Particle *pi, Particle *pj) {
    float dx = pi->position.x - pj->position.x;
    float dy = pi->position.y - pj->position.y;
    float dz = pi->position.z - pj->position.z;
    float dist2 = dx * dx + dy * dy + dz * dz;
    float h2 = deviceSettings.h * deviceSettings.h;

    if (dist2 > h2) {
        return 0.f;
    }

    float diff = h2 - dist2;
    return deviceSettings.d_kernel_coeff * diff * diff * diff;
}

// Smoothing kernel for pressure force updates
__device__ float3 pressureKernel(Particle *pi, Particle *pj) {
    float dx = pi->position.x - pj->position.x;
    float dy = pi->position.y - pj->position.y;
    float dz = pi->position.z - pj->position.z;
    float dist2 = dx * dx + dy * dy + dz * dz;

    if (dist2 > deviceSettings.h * deviceSettings.h) {
        return make_float3(0.f, 0.f, 0.f);
    }

    float dist = sqrtf(dist2);
    if (dist < EPS_F)
        return make_float3(0.f, 0.f, 0.f);

    float scale = (-deviceSettings.v_kernel_coeff) * (deviceSettings.h - dist) *
                  (deviceSettings.h - dist) / dist;

    return make_float3(dx * scale, dy * scale, dz * scale);
}

// Smoothing kernel for viscosity force updates
__device__ float viscosityKernel(Particle *pi, Particle *pj) {
    float dx = pi->position.x - pj->position.x;
    float dy = pi->position.y - pj->position.y;
    float dz = pi->position.z - pj->position.z;
    float dist = sqrtf(dx * dx + dy * dy + dz * dz);

    if ((dist > deviceSettings.h) || (dist < EPS_F)) {
        return 0.f;
    }

    return deviceSettings.v_kernel_coeff * (deviceSettings.h - dist);
}

// Kernels
__global__ void kernelAssignCellID(Particle *particles, int *metadata) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= deviceSettings.numParticles) {
        return;
    }

    Particle *particle = &particles[pIdx];
    int3 cell = getGridCell(particle->position);
    particle->cellID = flattenGridCoord(cell);

    // Update metadata array
    metadata[pIdx] = particle->cellID;
}

__global__ void kernelPopulateGrid(Particle *particles, int *neighborGrid) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= deviceSettings.numParticles) {
        return;
    }

    __shared__ Particle sharedParticles[MAX_THREADS_PER_BLOCK];
    sharedParticles[threadIdx.x] = particles[pIdx];

    __syncthreads();

    int myCellID = sharedParticles[threadIdx.x].cellID;
    int prevCellID = (pIdx == 0) ? 0
                     : (threadIdx.x == 0)
                         ? particles[pIdx - 1].cellID
                         : sharedParticles[threadIdx.x - 1].cellID;

    if (pIdx == 0 || myCellID != prevCellID) {
        neighborGrid[myCellID] = pIdx;
    }
}

__global__ void kernelUpdatePressureAndDensity(Particle *particles,
                                               int *neighborGrid) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int myChunkIdx = blockIdx.x;
    int totalChunks = gridDim.x;

    int startChunkIdx = max(myChunkIdx - (CHUNK_COUNT / 2), 0);

    if (myChunkIdx + (CHUNK_COUNT / 2) >= totalChunks) {
        startChunkIdx = max(0, totalChunks - CHUNK_COUNT);
    }

    int firstParticleIdx = startChunkIdx * MAX_THREADS_PER_BLOCK;

    if (pIdx >= deviceSettings.numParticles) {
        return;
    }

    // Shared array to store the particles related to this block
    __shared__ Particle myParticles[CHUNK_COUNT * MAX_THREADS_PER_BLOCK];

    for (int i = 0; i < CHUNK_COUNT; i++) {
        int particleToLoad = (firstParticleIdx + i * blockDim.x) + threadIdx.x;
        if (particleToLoad >= deviceSettings.numParticles)
            break;
        myParticles[particleToLoad - firstParticleIdx] =
            particles[particleToLoad];
    }

    __syncthreads();

    Particle *particle = NULL;
    if (CHUNK_COUNT > 0) {
        particle =
            &myParticles[(myChunkIdx - startChunkIdx) * MAX_THREADS_PER_BLOCK +
                         threadIdx.x];
    } else {
        particle = &particles[pIdx];
    }

    // Particle *particle =
    // &myParticles[(myChunkIdx - startChunkIdx) * MAX_THREADS_PER_BLOCK +
    // threadIdx.x];

    int3 cell = getGridCell(particle->position);
    particle->density = 0.f;

    // Update density based on neighbors
    for (int dz = -1; dz < 2; dz++) {
        int searchZ = cell.z + dz;
        if (searchZ < 0 || searchZ >= deviceSettings.numCellsPerDim)
            continue;
        for (int dy = -1; dy < 2; dy++) {
            int searchY = cell.y + dy;
            if (searchY < 0 || searchY >= deviceSettings.numCellsPerDim)
                continue;
            for (int dx = -1; dx < 2; dx++) {
                int searchX = cell.x + dx;
                if (searchX < 0 || searchX >= deviceSettings.numCellsPerDim)
                    continue;
                int neighborCellIdx =
                    flattenGridCoord(make_int3(searchX, searchY, searchZ));
                int neighborIdx = neighborGrid[neighborCellIdx];
                if (neighborIdx == -1)
                    continue;
                for (int i = neighborIdx; i < deviceSettings.numParticles;
                     i++) {
                    Particle *neighbor = NULL;

                    // Overall less branching in case of chunk count
                    // = 0 because all go to global mem

                    if ((i >= firstParticleIdx) &&
                        (i < firstParticleIdx +
                                 MAX_THREADS_PER_BLOCK * CHUNK_COUNT)) {
                        // Get particle from shared memory
                        neighbor = &myParticles[i - firstParticleIdx];
                    } else {
                        // Get particle from global memory
                        neighbor = &particles[i];
                    }
                    if (neighbor->cellID != neighborCellIdx)
                        break;
                    particle->density +=
                        MASS * densityKernel(particle, neighbor);
                }
            }
        }
    }

    particle->density = fmaxf(particle->density, EPS_F);
    // Update pressure using new density
    particle->pressure =
        fmaxf(0.f, GAS_CONSTANT * (particle->density - REST_DENSITY));

    // Write my particle back to global memory
    particles[pIdx] = *particle;
}

__global__ void kernelUpdateForces(Particle *particles, int *neighborGrid) {
    register int pIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int myChunkIdx = blockIdx.x;
    int totalChunks = gridDim.x;

    int startChunkIdx = max(myChunkIdx - (CHUNK_COUNT / 2), 0);

    if ((myChunkIdx + (CHUNK_COUNT / 2)) >= totalChunks) {
        startChunkIdx = max(0, totalChunks - CHUNK_COUNT);
    }

    int firstParticleIdx = startChunkIdx * MAX_THREADS_PER_BLOCK;

    if (pIdx >= deviceSettings.numParticles) {
        return;
    }

    // Shared array to store the particles related to this block
    __shared__ Particle myParticles[CHUNK_COUNT * MAX_THREADS_PER_BLOCK];

    for (int i = 0; i < CHUNK_COUNT; i++) {
        int particleToLoad = (firstParticleIdx + i * blockDim.x) + threadIdx.x;
        if (particleToLoad >= deviceSettings.numParticles)
            break;
        myParticles[particleToLoad - firstParticleIdx] =
            particles[particleToLoad];
    }

    __syncthreads();

    Particle *particle = NULL;
    if (CHUNK_COUNT > 0) {
        particle =
            &myParticles[(myChunkIdx - startChunkIdx) * MAX_THREADS_PER_BLOCK +
                         threadIdx.x];
    } else {
        particle = &particles[pIdx];
    }

    int3 cell = getGridCell(particle->position);
    particle->force.x = 0.f;
    particle->force.y = 0.f;
    particle->force.z = 0.f;

    // Update forces based on neighbors
    for (int dz = -1; dz < 2; dz++) {
        int searchZ = cell.z + dz;
        if (searchZ < 0 || searchZ >= deviceSettings.numCellsPerDim)
            continue;
        for (int dy = -1; dy < 2; dy++) {
            int searchY = cell.y + dy;
            if (searchY < 0 || searchY >= deviceSettings.numCellsPerDim)
                continue;
            for (int dx = -1; dx < 2; dx++) {
                int searchX = cell.x + dx;
                if (searchX < 0 || searchX >= deviceSettings.numCellsPerDim)
                    continue;
                int neighborCellIdx =
                    flattenGridCoord(make_int3(searchX, searchY, searchZ));
                int neighborIdx = neighborGrid[neighborCellIdx];
                if (neighborIdx == -1)
                    continue;
                for (int i = neighborIdx; i < deviceSettings.numParticles;
                     i++) {

                    Particle *neighbor = NULL;

                    if ((i >= firstParticleIdx) &&
                        (i < firstParticleIdx +
                                 MAX_THREADS_PER_BLOCK * CHUNK_COUNT)) {
                        // Get particle from shared memory
                        neighbor = &myParticles[i - firstParticleIdx];
                    } else {
                        // Get particle from global memory
                        neighbor = &particles[i];
                    }

                    if (neighbor->cellID != neighborCellIdx)
                        break;

                    // Calculate pressure force
                    float fPressure =
                        -MASS * (particle->pressure + neighbor->pressure) /
                        (2.f * neighbor->density);
                    float3 kern1 = pressureKernel(particle, neighbor);
                    kern1.x *= fPressure;
                    kern1.y *= fPressure;
                    kern1.z *= fPressure;
                    particle->force.x += kern1.x;
                    particle->force.y += kern1.y;
                    particle->force.z += kern1.z;

                    // Calculate viscosity force
                    float3 dv = make_float3(
                        neighbor->velocity.x - particle->velocity.x,
                        neighbor->velocity.y - particle->velocity.y,
                        neighbor->velocity.z - particle->velocity.z);
                    float fViscosity = VISCOSITY * MASS *
                                       viscosityKernel(particle, neighbor) /
                                       neighbor->density;
                    dv.x *= fViscosity;
                    dv.y *= fViscosity;
                    dv.z *= fViscosity;
                    particle->force.x += dv.x;
                    particle->force.y += dv.y;
                    particle->force.z += dv.z;
                }
            }
        }
    }

    // Write my particle back to global memory
    particles[pIdx] = *particle;
}

__global__ void kernelUpdatePositions(Particle *particles,
                                      float3 *devicePosition) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= deviceSettings.numParticles) {
        return;
    }

    Particle *particle = &particles[pIdx];
    float timestep = deviceSettings.timestep;

    if (!isfinite(particle->force.x) || !isfinite(particle->velocity.x)) {
        printf("Bad force/velocity at particle %d: fx=%f, vx=%f\n", pIdx,
               particle->force.x, particle->velocity.x);
    }
    if (!isfinite(particle->force.y) || !isfinite(particle->velocity.y)) {
        printf("Bad force/velocity at particle %d: fy=%f, vy=%f\n", pIdx,
               particle->force.y, particle->velocity.y);
    }
    if (!isfinite(particle->force.z) || !isfinite(particle->velocity.z)) {
        printf("Bad force/velocity at particle %d: fz=%f, vz=%f\n", pIdx,
               particle->force.z, particle->velocity.z);
    }

    particle->velocity.x += timestep * particle->force.x / particle->density;
    particle->velocity.y +=
        timestep * (particle->force.y / particle->density + GRAVITY);
    particle->velocity.z += timestep * particle->force.z / particle->density;

    particle->position.x += timestep * particle->velocity.x;
    particle->position.y += timestep * particle->velocity.y;
    particle->position.z += timestep * particle->velocity.z;

    // Handle boundary collisions
    if (particle->position.x < deviceSettings.h) {
        particle->position.x = deviceSettings.h;
        particle->velocity.x *= -ELASTICITY;
    } else if (particle->position.x >
               deviceSettings.boxDim - deviceSettings.h) {
        particle->position.x = deviceSettings.boxDim - deviceSettings.h;
        particle->velocity.x *= -ELASTICITY;
    }

    if (particle->position.y < deviceSettings.h) {
        particle->position.y = deviceSettings.h;
        particle->velocity.y *= -ELASTICITY;
    } else if (particle->position.y >
               deviceSettings.boxDim - deviceSettings.h) {
        particle->position.y = deviceSettings.boxDim - deviceSettings.h;
        particle->velocity.y *= -ELASTICITY;
    }

    if (particle->position.z < deviceSettings.h) {
        particle->position.z = deviceSettings.h;
        particle->velocity.z *= -ELASTICITY;
    } else if (particle->position.z >
               deviceSettings.boxDim - deviceSettings.h) {
        particle->position.z = deviceSettings.boxDim - deviceSettings.h;
        particle->velocity.z *= -ELASTICITY;
    }

    if (fabs(particle->velocity.x) < EPS_F) {
        particle->velocity.x = 0;
    }
    if (fabs(particle->velocity.y) < EPS_F) {
        particle->velocity.y = 0;
    }
    if (fabs(particle->velocity.z) < EPS_F) {
        particle->velocity.z = 0;
    }

    // Write updated positions
    devicePosition[pIdx] = particle->position;
}

// Reset the list heads
__global__ void kernelResetGrid(int *neighborGrid) {
    int listIdx = blockIdx.x + blockIdx.y * gridDim.y +
                  blockIdx.z * gridDim.z * gridDim.z;

    neighborGrid[listIdx] = -1;
}

// Induce velocity on mouse click
__global__ void kernelMoveParticles(Particle *particles, int *neighborGrid,
                                    int2 mouse_pos) {
    // Normalize the mouse positions to the box's size
    float x =
        ((float)(mouse_pos.x - BOX_MIN_X) / (float)(BOX_MAX_X - BOX_MIN_X)) *
        deviceSettings.boxDim;
    float y =
        ((float)(mouse_pos.y - BOX_MIN_Y) / (float)(BOX_MAX_Y - BOX_MIN_Y)) *
        deviceSettings.boxDim;
    float z = (float)threadIdx.x * deviceSettings.h;

    int3 cell = getGridCell(make_float3(x, y, z));
    cell.y = deviceSettings.numCellsPerDim - cell.y;

    for (int dy = -2; dy < 3; dy++) {
        int searchY = cell.y + dy;
        if (searchY < 0 || searchY >= deviceSettings.numCellsPerDim)
            continue;

        for (int dx = -2; dx < 3; dx++) {
            int searchX = cell.x + dx;
            if (searchX < 0 || searchX >= deviceSettings.numCellsPerDim)
                continue;
            int neighborCellIdx =
                flattenGridCoord(make_int3(searchX, searchY, cell.z));
            int neighborIdx = neighborGrid[neighborCellIdx];
            if (neighborIdx == -1)
                continue;
            for (int i = neighborIdx; i < deviceSettings.numParticles; i++) {
                Particle *neighbor = &particles[i];
                if (neighbor->cellID != neighborCellIdx)
                    break;
                if (dx != 0)
                    neighbor->velocity.x += (1.f / dx) * PUSH_STRENGTH;
                if (dy != 0)
                    neighbor->velocity.y += (1.f / dy) * PUSH_STRENGTH;
                if (dx == 0 && dy == 0)
                    neighbor->velocity.z -= PUSH_STRENGTH;
            }
        }
    }
}

// Class methods
Simulator::Simulator(Settings *settings) : settings(settings) {
    position = NULL;

    neighborGrid = NULL;
    particles = NULL;
}

Simulator::~Simulator() {
    if (position) {
        delete[] position;
    }

    // Free the grid on device
    if (neighborGrid != NULL) {
        hipFree(neighborGrid);
    }

    // Free the particles on device
    if (particles != NULL) {
        hipFree(particles);
        hipFree(devicePosition);
    }
}

const float3 *Simulator::getPosition() {
    return position;
}

void Simulator::setup() {
    // Initialize device data structures
    int neighborGridDim = settings->numCellsPerDim;
    hipMalloc(&neighborGrid, neighborGridDim * neighborGridDim *
                                  neighborGridDim * sizeof(int));
    hipMalloc(&particles, settings->numParticles * sizeof(Particle));
    hipMalloc(&devicePosition, settings->numParticles * sizeof(float3));
    hipMalloc(&metadata, settings->numParticles * sizeof(int));

    hipMemset(neighborGrid, -1,
               neighborGridDim * neighborGridDim * neighborGridDim *
                   sizeof(int));
    hipMemset(particles, 0, settings->numParticles * sizeof(Particle));

    // Initialize particle positions
    Particle *tmpParticles =
        (Particle *)malloc(sizeof(Particle) * settings->numParticles);

    position = (float3 *)malloc(sizeof(float3) * settings->numParticles);

    if (settings->randomInit) {
        for (size_t i = 0; i < settings->numParticles; i++) {
            float x = rand() / (float)RAND_MAX * (settings->boxDim - 2.f) + 1.f;
            float y = rand() / (float)RAND_MAX * (settings->boxDim - 2.f) + 1.f;
            float z = rand() / (float)RAND_MAX * (settings->boxDim - 2.f) + 1.f;

            tmpParticles[i] = Particle(make_float3(x, y, z));
        }
    } else {
        float spacing = 0.9f * settings->h;
        int nx = floor((settings->boxDim - 2 * settings->h) / spacing) + 1;
        int ny = nx, nz = nx;

        int count = 0;
        for (int x = 0; x < nx && count < settings->numParticles; x++) {
            for (int y = 0; y < ny && count < settings->numParticles; y++) {
                for (int z = 0; z < nz && count < settings->numParticles; z++) {
                    tmpParticles[count++] = Particle(make_float3(
                        settings->h + spacing * x, settings->h + spacing * y,
                        settings->h + spacing * z));
                }
            }
        }
    }

    // Copy initialized particles to device
    hipMemcpy(particles, tmpParticles,
               settings->numParticles * sizeof(Particle),
               hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(deviceSettings), settings, sizeof(Settings));
}

void Simulator::buildNeighborGrid() {
    // Assign cell id of each particle
    dim3 blockDim(MAX_THREADS_PER_BLOCK);
    dim3 gridDim((settings->numParticles + MAX_THREADS_PER_BLOCK - 1) /
                 MAX_THREADS_PER_BLOCK);

    kernelAssignCellID<<<gridDim, blockDim>>>(particles, metadata);

    // Sort particles array by cell id
    thrust::sort_by_key(thrust::device, metadata,
                        metadata + settings->numParticles, particles);

    // Populate neighborGrid
    kernelPopulateGrid<<<gridDim, blockDim>>>(particles, neighborGrid);
}

void Simulator::simulate() {
    // Build neighbor grid
    buildNeighborGrid();

    // Compute updates
    dim3 blockDim(MAX_THREADS_PER_BLOCK);
    dim3 gridDim((settings->numParticles + MAX_THREADS_PER_BLOCK - 1) /
                 MAX_THREADS_PER_BLOCK);

    kernelUpdatePressureAndDensity<<<gridDim, blockDim>>>(particles,
                                                          neighborGrid);
    kernelUpdateForces<<<gridDim, blockDim>>>(particles, neighborGrid);
    kernelUpdatePositions<<<gridDim, blockDim>>>(particles, devicePosition);
    hipDeviceSynchronize();

    // Copy updated positions to host
    hipMemcpy(position, devicePosition,
               sizeof(float3) * settings->numParticles, hipMemcpyDeviceToHost);

    // Handle mouse click
    if (mouseClicked) {
        dim3 blockDimClick(settings->numCellsPerDim);
        dim3 gridDimClick(1);

        kernelMoveParticles<<<gridDimClick, blockDimClick>>>(
            particles, neighborGrid, clickCoords);
        hipDeviceSynchronize();
        mouseClicked = false;
    }
}

void Simulator::simulateAndTime(Times *times) {
    // Build neighbor grid
    auto buildGridStart = std::chrono::steady_clock::now();

    buildNeighborGrid();

    times->buildGrid +=
        std::chrono::duration_cast<std::chrono::duration<double>>(
            std::chrono::steady_clock::now() - buildGridStart)
            .count();

    // Compute updates
    auto sphUpdateStart = std::chrono::steady_clock::now();

    dim3 blockDim(MAX_THREADS_PER_BLOCK);
    dim3 gridDim((settings->numParticles + MAX_THREADS_PER_BLOCK - 1) /
                 MAX_THREADS_PER_BLOCK);

    kernelUpdatePressureAndDensity<<<gridDim, blockDim>>>(particles,
                                                          neighborGrid);
    kernelUpdateForces<<<gridDim, blockDim>>>(particles, neighborGrid);
    kernelUpdatePositions<<<gridDim, blockDim>>>(particles, devicePosition);
    hipDeviceSynchronize();

    times->sphUpdate +=
        std::chrono::duration_cast<std::chrono::duration<double>>(
            std::chrono::steady_clock::now() - sphUpdateStart)
            .count();

    // Copy updated positions to host
    auto memcpyStart = std::chrono::steady_clock::now();

    hipMemcpy(position, devicePosition,
               sizeof(float3) * settings->numParticles, hipMemcpyDeviceToHost);

    times->memcpy += std::chrono::duration_cast<std::chrono::duration<double>>(
                         std::chrono::steady_clock::now() - memcpyStart)
                         .count();
    times->iters += 1;
}
